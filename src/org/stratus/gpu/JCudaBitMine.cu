#include "hip/hip_runtime.h"
#include<openssl/sha.h>
#include <stdio.h>
#include <string.h>



typedef struct block_header {
        unsigned char shainput[76];
        unsigned long nonce;
} block_header;

void byte_swap(unsigned char* data, int len) {
        int c;
        unsigned char tmp[len];

        c=0;
        while(c<len)
        {
                tmp[c] = data[len-(c+1)];
                c++;
        }

        c=0;
        while(c<len)
        {
                data[c] = tmp[c];
                c++;
        }
}

extern "C"

__global__ void inversehash(int n,char Input[],long Nonce[], char Target[],long Output)
{
    	int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i<n)
        {
        blockheader header;
        memcpy (header.shainput,Input,76);
        header.nonce = Nonce[i];
        unsigned char hash1[SHA256_DIGEST_LENGTH];
	unsigned char hash2[SHA256_DIGEST_LENGTH];
	SHA256_CTX sha256_pass1, sha256_pass2;
    	SHA256_Init(&sha256_pass1);
    	SHA256_Update(&sha256_pass1, (unsigned char*)header,76+sizeof(long));
    	SHA256_Final(hash1, &sha256_pass1);
    	SHA256_Init(&sha256_pass2);
    	SHA256_Update(&sha256_pass2,hash1,SHA256_DIGEST_LENGTH);
    	SHA256_Final(hash2, &sha256_pass2);
    	byte_swap(hash2, SHA256_DIGEST_LENGTH);
        if(strcmp (hash2,Target) == 0)
         {
              Output = i;
         }
        }

}
